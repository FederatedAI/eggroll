#include "hip/hip_runtime.h"

#ifdef __linux
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h> 
#include <sys/time.h>
#include <assert.h>
#include <math.h>

#include <gmp.h>
#include "cgbn/cgbn.h"
#include "../utility/support.h"

#include "../paillier.h"
#endif

#ifdef _WIN32	

#include<stdio.h>
#include "./cgbn.h"
#include "./cgbn_cuda.h"
#include "hip/hip_runtime.h"
#include "cuda_device_runtime_api.h"
#include ""
#include <assert.h>
#include "../paillier.h"

#endif

extern "C" 
{	
	#include "eggPaillier.h"
}
#include "vector_functions.h"

#define TPI 32
//#define BITS 4096
#define BITS 3072
#define INSTANCES 100000
#define BIT2BYTE(a) (a+7)>>3

typedef cgbn_context_t<TPI>         context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

typedef typename env_t::cgbn_t                bn_t;
typedef typename env_t::cgbn_local_t          bn_local_t;
typedef cgbn_mem_t<BITS> gpu_mpz;

typedef struct
{
	cgbn_mem_t<BITS> g;
	cgbn_mem_t<BITS> n;
	cgbn_mem_t<BITS> n2;
	cgbn_mem_t<BITS> max_int;
	cgbn_mem_t<BITS> sub_n_max;
	mp_bitcnt_t len;

}cgbn_public_key;

typedef struct
{
	cgbn_mem_t<BITS> p;
	cgbn_mem_t<BITS> q;
	cgbn_mem_t<BITS> p2;
	cgbn_mem_t<BITS> q2;
	cgbn_mem_t<BITS> qinver;
	cgbn_mem_t<BITS> hp;
	cgbn_mem_t<BITS> hq;
	mp_bitcnt_t len;

}cgbn_private_key;


typedef struct
{
	gpu_mpz* d_field;
	uint64_t* d_exp;
	short* d_sign;
	int row;
	int col;
	int ifmal;
	int ifenc;
}gpu_manager;

extern "C" 
{
#define MASK64_EXP  0x7FF0000000000000
#define MASK64_MANT 0x000FFFFFFFFFFFFF
#define MASK64_ONE  0x0010000000000000

#define MASK32_EXP  0x7F800000
#define MASK32_MANT 0x007FFFFF
#define MASK32_ONE  0x00800000

	static void initManager_d(mpz_manager* target, int col, int row, int bits)
	{
		int i;
		target->col = col;
		target->row = row;
		target->field = (mpz_t*)malloc(sizeof(mpz_t) * col * row);
		target->exp = (int64_t*)malloc(sizeof(int64_t) * col * row);
		for (i = 0; i < col * row; i++)
		{
			mpz_init(target->field[i]);
			mpz_setbit(target->field[i], bits);
			target->exp[i] = 0;
		}
		target->ifenc = 0;
		target->ifmal = 1;
	}

	void store2dev(cgbn_mem_t<BITS>* address, mpz_t z) {
		size_t words;
		if (mpz_sizeinbase(z, 2) > BITS) {
			printf("error mpz_sizeinbase:%d\n", mpz_sizeinbase(z, 2));
			exit(1);
		}
		mpz_export((uint32_t*)address, &words, -1, sizeof(uint32_t), 0, 0, z);

		while (words < (BITS + 31) / 32)
		{
			((uint32_t*)address)[words++] = 0;
		}
	}

	void store2gmp(mpz_t z, cgbn_mem_t<BITS>* address) {
		mpz_import(z, (BITS + 31) / 32, -1, sizeof(uint32_t), 0, 0, (uint32_t*)address);
	}

	void initGpuManager(gpu_manager* d_target, int col, int row)
	{
		/* malloc */
		gpu_manager* h_contain = (gpu_manager*)malloc(sizeof(gpu_manager));

		h_contain->col = col;
		h_contain->row = row;
		h_contain->ifenc = 0; 
		h_contain->ifmal = 1;
		hipMalloc((void**)& h_contain->d_field, sizeof(gpu_mpz) * col * row);
		hipMalloc((void**)& h_contain->d_exp, sizeof(uint64_t) * col * row);
		hipMemset(h_contain->d_field, 0, sizeof(gpu_mpz) * col * row);
		hipMemset(h_contain->d_exp, 0, sizeof(uint64_t) * col * row);

		/* init device gpu_manager */
		hipMemcpy(d_target, h_contain, sizeof(gpu_manager), hipMemcpyHostToDevice);
		free(h_contain);
	}

	/* ========================= gpu paillier kernel ==================== */

	__device__ void eggroll_mulc_dev(env_t* env, env_t::cgbn_t* ciphertext2,
		env_t::cgbn_t* ciphertext1, env_t::cgbn_t* _const, env_t::cgbn_t* n2,
		env_t::cgbn_t* n, env_t::cgbn_t* sub_n_max, env_t::cgbn_t* nc, 
		env_t::cgbn_t* ns)
	{
		if (cgbn_compare(*env, *_const, *sub_n_max) >= 0)
		{
			cgbn_modular_inverse(*env, *nc, *ciphertext1, *n2);
			cgbn_sub(*env, *ns, *n, *_const);
			cgbn_modular_power(*env, *ciphertext2, *nc, *ns, *n2);
		}
		else
		{
			cgbn_modular_power(*env, *ciphertext2, *ciphertext1, *_const, *n2);
		}
	}

	__device__ uint64_t eggroll_alinexp(env_t* env, uint64_t exp1, uint64_t exp2, 
		env_t::cgbn_t* cipher1, env_t::cgbn_t* cipher2, env_t::cgbn_t* n2,
		env_t::cgbn_t* n, env_t::cgbn_t* sub_n_max, env_t::cgbn_t* _f, 
		env_t::cgbn_t* nc, env_t::cgbn_t* ns)
	{
		if (exp1 < exp2)
		{
			//pow(base, exp2 - exp1)
			cgbn_set_ui32(*env, *_f, 1);
			for (int i = 0; i < exp2 - exp1; i++)
			{
				cgbn_mul_ui32(*env, *_f, *_f, (uint32_t)BASE);
			}

			cgbn_rem(*env, *_f, *_f, *n);
			eggroll_mulc_dev(env, cipher1, cipher1, _f, n2, n, sub_n_max, nc, ns);

			return exp2;
		}
		else
		{
			//change cipher2
			//mpz_set_ui(_f, (int64_t)pow(BASE, exp1 - exp2));
			cgbn_set_ui32(*env, *_f, 1);
			for (int i = 0; i < exp1 - exp2; i++)
			{
				cgbn_mul_ui32(*env, *_f, *_f, (uint32_t)BASE);
			}

			cgbn_rem(*env, *_f, *_f, *n);
			eggroll_mulc_dev(env, cipher2, cipher2, _f, n2, n, sub_n_max, nc, ns);
			return exp1;
		}
	}

	__global__ void dev_encrpyt_kernel(int col, int row, cgbn_public_key* d_pub,
		gpu_manager* d_mng, gpu_manager* d_mng_enc, cgbn_error_report_t* report)
	{

		int n32 = BIT2BYTE(BITS) / 4;

		int id = blockIdx.x * blockDim.x + threadIdx.x;
		int team = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;

		if (team >= col * row)
			return;
	/*	printf("[encrypt] %d\n", team);*/



		context_t      bn_context(cgbn_report_monitor, report, team);   // construct a context
		env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math

		env_t::cgbn_t  n, n2, plain, sub_n_max, max_int, n_plain, n_cipher, cipher;

		cgbn_load(bn_env, n, &d_pub->n);
		cgbn_load(bn_env, n2, &d_pub->n2);
		cgbn_load(bn_env, max_int, &d_pub->max_int);
		cgbn_load(bn_env, sub_n_max, &d_pub->sub_n_max);
		cgbn_load(bn_env, plain, d_mng->d_field + team);

		if (cgbn_compare(bn_env, plain, sub_n_max) >= 0 &&
			cgbn_compare(bn_env, plain, n) < 0)
		{
			// Very large plaintext, take a sneaky shortcut using inverses
			cgbn_sub(bn_env, n_plain, n, plain);
			cgbn_mul(bn_env, n_cipher, n, n_plain);

			cgbn_add_ui32(bn_env, n_cipher, n_cipher, 1);
			cgbn_rem(bn_env, n_cipher, n_cipher, n2);
			cgbn_modular_inverse(bn_env, cipher, n_cipher, n2);

		}
		else {
			cgbn_mul(bn_env, cipher, n, plain);
			cgbn_add_ui32(bn_env, cipher, cipher, 1);
			cgbn_rem(bn_env, cipher, cipher, n2);
		}
		/* set filed */
		cgbn_store(bn_env, d_mng_enc->d_field + team, cipher);   // store r into sum
		/* set filed */
		d_mng_enc->d_exp[team] = d_mng->d_exp[team];
		d_mng_enc->ifenc = 1;


		//if (id == 0)
		//{
		//	for (int i = 0; i < n32; i++)
		//	{
		//		printf("%d ", plain);
		//	}
		//}
	}

	__global__ void dev_dotadd_kernel(int col, int row, cgbn_public_key* d_pub,
		gpu_manager* d_mat1, gpu_manager* d_mat2, gpu_manager* d_res,
		cgbn_error_report_t* report)
	{

		int n32 = BIT2BYTE(BITS) / 4;

		int id = blockIdx.x * blockDim.x + threadIdx.x;
		int team = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;

		if (team >= col * row)
			return;

		//printf("[encrypt] %d\n", team);


		__shared__ uint64_t exp1[4];
		__shared__ uint64_t exp2[4];
		__shared__ uint64_t power[4];

		context_t      bn_context(cgbn_report_monitor, report, team);   // construct a context
		env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math

		///* variant */
		env_t::cgbn_t  r, n, n2, sub_n_max, cipher1, cipher2;
		env_t::cgbn_t  p, f, nc, ns;

		exp1[team % 4] = d_mat1->d_exp[team];
		exp2[team % 4] = d_mat2->d_exp[team];

		cgbn_load(bn_env, n2, &d_pub->n2);
		cgbn_load(bn_env, n, &d_pub->n);
		cgbn_load(bn_env, sub_n_max, &d_pub->sub_n_max);
		cgbn_load(bn_env, cipher1, d_mat1->d_field + team);
		cgbn_load(bn_env, cipher2, d_mat2->d_field + team);

		if (exp1[team % 4] == exp2[team % 4])
		{

			//printf("exp1[l] = %ld exp2[r] = %ld\n", exp1[team % 4], exp2[team % 4]);

			cgbn_mul(bn_env, r, cipher1, cipher2);
			cgbn_rem(bn_env, r, r, n2);
			d_res->d_exp[team % 4] = exp1[team % 4];
		}
		else
		{
			printf("exp1[l] = %ld exp2[r] = %ld\n", exp1[team % 4], exp2[team % 4]);

			//aline cipher
			d_res->d_exp[team % 4] = eggroll_alinexp(&bn_env, exp1[team % 4],
				exp2[team % 4], &cipher1, &cipher2, &n2, &n, &sub_n_max, &f, &nc, &ns);

			/* eggroll_add */
			cgbn_mul(bn_env, r, cipher1, cipher2);
			cgbn_rem(bn_env, r, r, n2);
		}
		cgbn_store(bn_env, d_res->d_field + team, r);

	}

	/* gpu mpz dot mul*/
	__global__ void dev_dotmul_kernel(int col, int row, cgbn_public_key* d_pub,
		gpu_manager* d_mat1, gpu_manager* d_mat2, gpu_manager* d_res,
		cgbn_error_report_t* report)
	{
#if 0
		int n32 = BIT2BYTE(BITS) / 4;

		int id = blockIdx.x * blockDim.x + threadIdx.x;
		int team = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;

		if (team >= col * row)
			return;

		context_t      bn_context(cgbn_report_monitor, report, team);   // construct a context
		env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math

		/* variant */
		env_t::cgbn_t  sub_n_max, n, n2;
		env_t::cgbn_t  nc, ns, cmp, cipher, constant, res;

		cgbn_load(bn_env, sub_n_max, &d_pub->sub_n_max);
		cgbn_load(bn_env, n, &d_pub->n);
		cgbn_load(bn_env, n2, &d_pub->n2);
		cgbn_load(bn_env, cipher, d_mat1->d_field + team);
		cgbn_load(bn_env, constant, d_mat2->d_field + team);
		cgbn_load(bn_env, res, d_res->d_field + team);

		/* todo judgement */

		eggroll_mulc_dev(&bn_env, &res, &cipher, 
			&constant, &n2, &n, &sub_n_max, &nc, &ns);
		
		d_res->d_exp[team] = d_mat1->d_exp[team] + d_mat2->d_exp[team];
		cgbn_store(bn_env, d_res->d_field + team, res);   // store r into sum

#endif
	}

	/* gpu mpz mat mul*/
	__global__ void dev_matmul_c_eql_kernel(int col1, int row1, int col2, int row2,
		cgbn_public_key* d_pub, gpu_manager* d_mat1, gpu_manager* d_mat2, gpu_manager* d_res,
		cgbn_error_report_t* report)
	{
#if 1
		int n32 = BIT2BYTE(BITS) / 4;

		int id = blockIdx.x * blockDim.x + threadIdx.x;
		int team = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;
		

		//launch col1 num team		
		if (team >= row1)
			return;

		__shared__ uint64_t _exp1[4];
		__shared__ uint64_t _exp2[4];
		_exp1[team % 4] = 0;
		_exp2[team % 4] = 0;

		context_t      bn_context(cgbn_report_monitor, report, team);   // construct a context
		env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math

		/* variant */
		env_t::cgbn_t  sub_n_max, n, n2;
		env_t::cgbn_t  mul_tmp, add_tmp;
		env_t::cgbn_t  r, f, nc, ns, cipher, constant;

		cgbn_load(bn_env, sub_n_max, &d_pub->sub_n_max);
		cgbn_load(bn_env, n, &d_pub->n);
		cgbn_load(bn_env, n2, &d_pub->n2);
		
		cgbn_set_ui32(bn_env, add_tmp, 1);

		/* todo judgement */

		/* travel col1 */
		for (int i = 0; i < col1; i++)
		{
			/* locate offset */
			cgbn_load(bn_env, cipher, d_mat1->d_field + (i + team * col1));
			cgbn_load(bn_env, constant, d_mat2->d_field + i);

			/* do mul => [mul_tmp, _exp1] */
			eggroll_mulc_dev(&bn_env, &mul_tmp, &cipher, 
				&constant, &n2, &n, &sub_n_max, &nc, &ns);

			_exp1[team % 4] =  d_mat1->d_exp[i + team * col1] + d_mat2->d_exp[i];
			
			
			/* do add ==> [add_tmp, _exp2] */
			if (_exp1[team % 4] == _exp2[team % 4])
			{
				cgbn_mul(bn_env, r, add_tmp, mul_tmp);
				cgbn_rem(bn_env, r, r, n2);
				_exp2[team % 4] = _exp1[team % 4];
			}
			else
			{
				//aline cipher
				_exp2[team % 4] = eggroll_alinexp(&bn_env, _exp1[team % 4],
					_exp2[team % 4], &mul_tmp, &add_tmp, &n2,
					&n, &sub_n_max, &f, &nc, &ns);

				/* eggroll_add */
				cgbn_mul(bn_env, r, add_tmp, mul_tmp);
				cgbn_rem(bn_env, r, r, n2);
			}
			cgbn_add_ui32(bn_env, add_tmp, r, 0);
		}
		//test
		cgbn_store(bn_env, d_res->d_field + team, add_tmp);   // store r into sum
		d_res->d_exp[team] = _exp2[team % 4];
#endif
	}
	
	/* gpu_mng decrpyt */
	__global__ void dev_decrypt_kernel(int col, int row, cgbn_public_key* d_pub,
		cgbn_private_key* d_priv,gpu_manager* d_cipher, gpu_manager* d_plain,
		cgbn_error_report_t* report)
	{
#if 1
		int n32 = BIT2BYTE(BITS) / 4;

		int id = blockIdx.x * blockDim.x + threadIdx.x;
		int team = (blockIdx.x * blockDim.x + threadIdx.x) / TPI;

		if (team >= col * row)
			return;

		context_t      bn_context(cgbn_report_monitor, report, team);   // construct a context
		env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math

		/* variant */
		env_t::cgbn_t cipher, plain;
		env_t::cgbn_t b_cipher;
		env_t::cgbn_t n, hp, p, p2, hq, q, q2, qinv;
		env_t::cgbn_t r, s, u, mp, mq;

		//env_t::cgbn_wide_t r;
		cgbn_load(bn_env, n, &d_pub->n);
		cgbn_load(bn_env, hp, &d_priv->hp);
		cgbn_load(bn_env, p, &d_priv->p);
		cgbn_load(bn_env, p2, &d_priv->p2);
		cgbn_load(bn_env, hq, &d_priv->hq);
		cgbn_load(bn_env, q, &d_priv->q);
		cgbn_load(bn_env, q2, &d_priv->q2);
		cgbn_load(bn_env, qinv, &d_priv->qinver);
		cgbn_load(bn_env, cipher, d_cipher->d_field + team);
		cgbn_load(bn_env, plain, d_plain->d_field + team);

		//mp
		cgbn_sub_ui32(bn_env, r, p, 1);

		if (cgbn_compare(bn_env, cipher, p2) >= 0)
		{
			/* base > moder */
			cgbn_rem(bn_env, b_cipher, cipher, p2);
			cgbn_modular_power(bn_env, s, b_cipher, r, p2);
		}
		else
		{
			/* base < moder */
			cgbn_modular_power(bn_env, s, cipher, r, p2);
		}

		cgbn_sub_ui32(bn_env, r, s, 1);
		cgbn_div(bn_env, s, r, p);
		cgbn_mul(bn_env, r, s, hp);
		cgbn_rem(bn_env, mp, r, p);
		
		//mq
		cgbn_sub_ui32(bn_env, r, q, 1);
		if (cgbn_compare(bn_env, cipher, q2) >= 0)
		{
			/* base > moder */
			cgbn_rem(bn_env, b_cipher, cipher, q2);
			cgbn_modular_power(bn_env, s, b_cipher, r, q2);
		}
		else
		{
			/* base < moder */
			cgbn_modular_power(bn_env, s, cipher, r, q2);
		}
		cgbn_sub_ui32(bn_env, r, s, 1);
		cgbn_div(bn_env, s, r, q);
		cgbn_mul(bn_env, r, s, hq);
		cgbn_rem(bn_env, mq, r, q);

		cgbn_sub(bn_env, r, mp, mq);
		cgbn_mul(bn_env, s, r, qinv);
		cgbn_rem(bn_env, u, s, p);

		cgbn_mul(bn_env, r, u, q);
		cgbn_add(bn_env, s, mq, r);
		cgbn_rem(bn_env, plain, s, n);

		d_plain->d_exp[team] = d_cipher->d_exp[team];
		cgbn_store(bn_env, d_plain->d_field + team, plain);   // store r into sum

#endif
	}


	/* ========================= gpu call function ===================== */

	void gpu_init_pub(eggroll_public_key* h_pub, void** d_pub)
	{
		////hipSetDevice(1);

		//printf("[host pub.g] = %d\n", h_pub->g->_mp_d[0]);
		//printf("[host pub.n] = %d\n", h_pub->n->_mp_d[0]);
		//printf("[host pub.n2] = %d\n", h_pub->n2->_mp_d[0]);
		//printf("[host pub.max] = %d\n", h_pub->max_int->_mp_d[0]);

		/* ember value */
		cgbn_public_key dev_pub_key;
		store2dev(&dev_pub_key.g, h_pub->g);
		store2dev(&dev_pub_key.n, h_pub->n);
		store2dev(&dev_pub_key.n2, h_pub->n2);
		store2dev(&dev_pub_key.max_int, h_pub->max_int);
		store2dev(&dev_pub_key.sub_n_max, h_pub->sub_n_max);
		dev_pub_key.len = BITS;

		/* pub key host2dev */
		void* tmp_pub;
		hipMalloc((void**)& tmp_pub, sizeof(cgbn_public_key));
		hipMemcpy(tmp_pub, &dev_pub_key, sizeof(cgbn_public_key),
			hipMemcpyHostToDevice);
		//printf("[tmp]: d_pub = %0x\n", tmp_pub);

		/* copy gpu adress*/
		memcpy(d_pub, &tmp_pub, sizeof(void*));
		//printf("[tmp after]: d_pub = %0x\n", *d_pub);

	}

	void gpu_init_priv(eggroll_private_key* h_priv, void** d_priv)
	{
		//printf("[host pub.g] = %d\n", h_priv->hp->_mp_d[0]);

		////hipSetDevice(1);

		/* ember value */
		cgbn_private_key cgbn_priv;
		store2dev(&cgbn_priv.hp, h_priv->hp);
		store2dev(&cgbn_priv.hq, h_priv->hq);
		store2dev(&cgbn_priv.p, h_priv->p);
		store2dev(&cgbn_priv.q, h_priv->q);
		store2dev(&cgbn_priv.p2, h_priv->p2);
		store2dev(&cgbn_priv.q2, h_priv->q2);
		store2dev(&cgbn_priv.qinver, h_priv->qinver);
		cgbn_priv.len = BITS;

		/* priv key host2dev */
		void* tmp_priv;
		hipMalloc((void**)& tmp_priv, sizeof(cgbn_private_key));
		hipMemcpy(tmp_priv, &cgbn_priv, sizeof(cgbn_private_key),
			hipMemcpyHostToDevice);
		
		/* copy gpu adress*/
		memcpy(d_priv, &tmp_priv, sizeof(void*));
	}

	void load_devPub(eggroll_public_key* h_pub, void** d_pub)
	{
		/* pub key Dev2host */
		cgbn_public_key h_cgbn_pub;
		hipMemcpy(&h_cgbn_pub, *d_pub, sizeof(cgbn_public_key),
			hipMemcpyDeviceToHost);

		/* load value */
		store2gmp(h_pub->g, &h_cgbn_pub.g);
		store2gmp(h_pub->n, &h_cgbn_pub.n);
		store2gmp(h_pub->n2, &h_cgbn_pub.n2);
		store2gmp(h_pub->max_int, &h_cgbn_pub.max_int);
		store2gmp(h_pub->sub_n_max, &h_cgbn_pub.sub_n_max);
		h_pub->len = BITS;

	}

	void load_devPriv(eggroll_private_key* h_priv, void** d_priv)
	{
		cgbn_private_key h_cgbn_priv;
		hipMemcpy(&h_cgbn_priv, *d_priv, sizeof(cgbn_public_key),
			hipMemcpyDeviceToHost);

		/* ember value */
		cgbn_private_key cgbn_priv;
		store2gmp(h_priv->hp, &cgbn_priv.hp);
		store2gmp(h_priv->hq, &cgbn_priv.hq);
		store2gmp(h_priv->p, &cgbn_priv.p);
		store2gmp(h_priv->q, &cgbn_priv.q);
		store2gmp(h_priv->p2, &cgbn_priv.p2);
		store2gmp(h_priv->q2, &cgbn_priv.q2);
		store2gmp(h_priv->qinver, &cgbn_priv.qinver);
		cgbn_priv.len = BITS;
	}

	void gpu_init_mpz(int col, int row, mpz_manager* c_mng, void** d_buf)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int count = col * row;
		mpz_t tmp;
		mpz_init(tmp);

		///* init gpu_mng*/
		gpu_manager* d_mng;
		hipMalloc((void**)& d_mng, sizeof(gpu_manager));
		//initGpuManager(d_mng, col, row);

		/* malloc contain */
		gpu_manager* h_contain = (gpu_manager*)malloc(sizeof(gpu_manager));
		
		h_contain->col = col;
		h_contain->row = row;
		h_contain->ifenc = 0;
		h_contain->ifmal = 1;
		
		hipMalloc((void**)& h_contain->d_field, sizeof(gpu_mpz) * col * row);
		hipMalloc((void**)& h_contain->d_exp, sizeof(uint64_t) * col * row);

		/* field host2dev */
		for (i = 0; i < col * row; i++)
		{
			//gmp_printf("[XXXXXgpu init ] : %Zd\n", c_mng->field[i]);
			hipMemcpy(h_contain->d_field[i]._limbs, c_mng->field[i]->_mp_d,
				nByte, hipMemcpyHostToDevice);
		}
		/* exp host2dev */
		hipMemcpy(h_contain->d_exp, c_mng->exp,
			sizeof(uint64_t) * col * row, hipMemcpyHostToDevice);

		/* contain host2dev*/
		hipMemcpy(d_mng, h_contain, sizeof(gpu_manager), hipMemcpyHostToDevice);

		/* set return */
		memcpy(d_buf, &d_mng, sizeof(void*));

		/* free contain*/
		free(h_contain);

		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_buf, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * col * row);
			
			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * col * row, hipMemcpyDeviceToHost);

			for (i = 0; i < col * row; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu init ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu init ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu init++] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu init ] exp = %ld\n", _e[i]);

			}


		}

	}

	void gpu_init_mpz_d(int col, int row, float64_t* c_buf, void** d_buf)
	{

		//hipSetDevice(1);
		printf("=======================\n");

		int i, j;
		int nByte = BIT2BYTE(BITS);
		int count = col * row;
		mpz_t tmp;
		mpz_init(tmp);

		/* malloc gpu_mng*/
		gpu_manager* d_mng = (gpu_manager*)malloc(sizeof(gpu_manager));
		initGpuManager(d_mng, col, row);

		/* init */
		//dev_encode_kernel_d<<<>>>()

		/* init field*/
		memcpy(d_buf, &d_mng, sizeof(void*));

	}
	
	void gpu_encrypt(void** d_pub, void** d_buf, void** d_enc_buf)
	{
		////hipSetDevice(1);
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col, row;

		cgbn_error_report_t* report;
		cgbn_error_report_alloc(&report);

		/* load d_buf */
		gpu_manager* h_buf;
		hipHostMalloc((void**)& h_buf, sizeof(gpu_manager));
		hipMemcpy(h_buf, *d_buf, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col = h_buf->col;
		row = h_buf->row;
		
		///* malloc d_enc_buf */
		gpu_manager* _enc_buf;
		hipMalloc((void**)& _enc_buf, sizeof(gpu_manager));
		initGpuManager(_enc_buf, col, row);

		/* calc thread num*/
		int block = (col * row + 3) / 4;
		int thread = 128;
		dev_encrpyt_kernel << <block, thread >> > (col, row, (cgbn_public_key*)(*d_pub),
			(gpu_manager*)(*d_buf), _enc_buf, report);
		hipDeviceSynchronize();

		memcpy(d_enc_buf, &_enc_buf, sizeof(void*));

		hipHostFree(h_buf);

		/* test */

		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_enc_buf, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * col * row);

			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * col * row, hipMemcpyDeviceToHost);

			for (i = 0; i < col * row; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu encrypt ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu encrypt ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu encrypt++] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu encrypt ] exp = %ld\n", _e[i]);

			}
		}
	}

	int gpu_dotadd(void** d_pub, void** d_mat1, void** d_mat2, void** d_res)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col1, row1, enc1;
		int col2, row2, enc2;

		cgbn_error_report_t* report;
		cgbn_error_report_alloc(&report);

		/* load d_mat1 */
		gpu_manager* h_mat1;
		hipHostMalloc((void**)& h_mat1, sizeof(gpu_manager));
		hipMemcpy(h_mat1, *d_mat1, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col1 = h_mat1->col;
		row1 = h_mat1->row;
		enc1 = h_mat1->ifenc;

		/* load d_mat2 */
		gpu_manager* h_mat2;
		hipHostMalloc((void**)& h_mat2, sizeof(gpu_manager));
		hipMemcpy(h_mat2, *d_mat2, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col2 = h_mat2->col;
		row2 = h_mat2->row;
		enc2 = h_mat2->ifenc;

		if (col1 != col2 || row1 != row2
			|| enc1 != 1 || enc2 != 1)
		{
			printf("[c1 r1 enc1] = [%d %d %d]\n",
				h_mat1->col, h_mat1->row, h_mat1->ifenc);
			printf("[c2 r2 enc21] = [%d %d %d]\n",
				h_mat2->col, h_mat2->row, h_mat2->ifenc);

			return -1;
		}

		/* malloc d_enc_buf */
		gpu_manager* _d_res;
		hipMalloc((void**)& _d_res, sizeof(gpu_manager));
		initGpuManager(_d_res, col1, row1);

		///* calc thread num */
		int block = (col1 * row1 + 3) / 4;
		int thread = 128;

		dev_dotadd_kernel << <block, thread >> > (col1, row1, (cgbn_public_key*)(*d_pub),
			(gpu_manager*)(*d_mat1), (gpu_manager*)(*d_mat2), _d_res, report);
		hipDeviceSynchronize();
		
		memcpy(d_res, &_d_res, sizeof(void*));

		hipHostFree(h_mat1);
		hipHostFree(h_mat2);


		//* test */
		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_mat1, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * col1 * row1);

			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * col1 * row1, hipMemcpyDeviceToHost);

			for (i = 0; i < col1 * row1; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu dotadd]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu dotadd ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu dotadd] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu dotadd] exp = %ld\n", _e[i]);

			}


		}

		return 1;

	}

	int gpu_dotmul(void** d_pub, void** d_mat1, void** d_mat2, void** d_res)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col1, row1, enc1;
		int col2, row2, enc2;

		cgbn_error_report_t* report;
		cgbn_error_report_alloc(&report);

		/* load d_mat1 */
		gpu_manager* h_mat1;
		hipHostMalloc((void**)& h_mat1, sizeof(gpu_manager));
		hipMemcpy(h_mat1, *d_mat1, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col1 = h_mat1->col;
		row1 = h_mat1->row;
		enc1 = h_mat1->ifenc;

		/* load d_mat2 */
		gpu_manager* h_mat2;
		hipHostMalloc((void**)& h_mat2, sizeof(gpu_manager));
		hipMemcpy(h_mat2, *d_mat2, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col2 = h_mat2->col;
		row2 = h_mat2->row;
		enc2 = h_mat2->ifenc;

		if (col1 != col2 || row1 != row2
			|| enc1 != 1 || enc2 != 0)
		{
			printf("[c1 r1 enc1] = [%d %d %d]\n",
				h_mat1->col, h_mat1->row, h_mat1->ifenc);
			printf("[c2 r2 enc21] = [%d %d %d]\n",
				h_mat2->col, h_mat2->row, h_mat2->ifenc);

			return -1;
		}

		/* malloc d_enc_buf */
		gpu_manager* _d_res;
		hipMalloc((void**)& _d_res, sizeof(gpu_manager));
		initGpuManager(_d_res, col1, row1);

		/* calc thread num */
		int block = (col1 * row1 + 3) / 4;
		int thread = 128;
		dev_dotmul_kernel <<<block, thread >>> (col1, row1, (cgbn_public_key*)(*d_pub),
			(gpu_manager*)(*d_mat1), (gpu_manager*)(*d_mat2), _d_res, report);
		hipDeviceSynchronize();

		memcpy(d_res, &_d_res, sizeof(void*));

		hipHostFree(h_mat1);
		hipHostFree(h_mat2);


		//* test */
		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_res, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * col1 * row1);

			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * col1 * row1, hipMemcpyDeviceToHost);

			for (i = 0; i < col1 * row1; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu dotmul ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu dotmul ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu dotmul++] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu dotmul ] exp = %ld\n", _e[i]);

			}

		}
		return 1;

	}

	int gpu_matmul_c_eql(void** d_pub, void** d_mat1, void** d_mat2, void** d_res)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col1, row1, enc1;
		int col2, row2, enc2;

		cgbn_error_report_t* report;
		cgbn_error_report_alloc(&report);

		/* load d_mat1 */
		gpu_manager* h_mat1;
		hipHostMalloc((void**)& h_mat1, sizeof(gpu_manager));
		hipMemcpy(h_mat1, *d_mat1, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col1 = h_mat1->col;
		row1 = h_mat1->row;
		enc1 = h_mat1->ifenc;

		/* load d_mat2 */
		gpu_manager* h_mat2;
		hipHostMalloc((void**)& h_mat2, sizeof(gpu_manager));
		hipMemcpy(h_mat2, *d_mat2, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col2 = h_mat2->col;
		row2 = h_mat2->row;
		enc2 = h_mat2->ifenc;

		/* mat mul require row_eql*/
		if (col1 != col2 || enc1 != 1 || enc2 != 0)
		{
			printf("[c1 r1 enc1] = [%d %d %d]\n",
				h_mat1->col, h_mat1->row, h_mat1->ifenc);
			printf("[c2 r2 enc21] = [%d %d %d]\n",
				h_mat2->col, h_mat2->row, h_mat2->ifenc);

			return -1;
		}

		/* malloc d_enc_buf */
		gpu_manager* _d_res;
		hipMalloc((void**)& _d_res, sizeof(gpu_manager));
		initGpuManager(_d_res, row1, row2);

		/* calc thread num */
		int block = (row1 + 3) / 4;
		int thread = 128;
		dev_matmul_c_eql_kernel <<<block, thread >>> (col1, row1, col2, row2, (cgbn_public_key*)(*d_pub),
			(gpu_manager*)(*d_mat1), (gpu_manager*)(*d_mat2), _d_res, report);
		hipDeviceSynchronize();

		memcpy(d_res, &_d_res, sizeof(void*));

		hipHostFree(h_mat1);
		hipHostFree(h_mat2);


		//* test */
		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_res, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * row1);

			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * row1, hipMemcpyDeviceToHost);

			for (i = 0; i < row1; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu dotmul ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu dotmul ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu dotmul++] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu dotmul ] exp = %ld\n", _e[i]);

			}

		}
		return 1;

	}

	void gpu_decrypt(void** d_pub, void** d_priv, void** d_cipher,
		void** d_plain)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col, row, enc;

		cgbn_error_report_t* report;
		cgbn_error_report_alloc(&report);

		/* load d_cipher */
		gpu_manager* h_cipher;
		hipHostMalloc((void**)& h_cipher, sizeof(gpu_manager));
		hipMemcpy(h_cipher, *d_cipher, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col = h_cipher->col;
		row = h_cipher->row;
		enc = h_cipher->ifenc;

		///* malloc d_enc_buf */
		gpu_manager* _d_plain;
		hipMalloc((void**)& _d_plain, sizeof(gpu_manager));
		initGpuManager(_d_plain, col, row);

		/* calc thread num */
		int block = (row * col + 3) / 4;
		int thread = 128;
		dev_decrypt_kernel << <block, thread >> > (col, row, (cgbn_public_key*)(*d_pub),
			(cgbn_private_key*)(*d_priv), (gpu_manager*)(*d_cipher), _d_plain, report);
		hipDeviceSynchronize();

		memcpy(d_plain, &_d_plain, sizeof(void*));

		hipHostFree(h_cipher);

		/* test */
		if (0)
		{
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));
			hipMemcpy(_tmp, *d_plain, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t* _e = (uint64_t*)malloc(sizeof(uint64_t) * col * row);

			/* exp dev2host*/
			hipMemcpy(_e, _tmp->d_exp, sizeof(uint64_t) * col * row, hipMemcpyDeviceToHost);

			for (i = 0; i < col * row; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);
				/*for (int i = 0; i < _t->_mp_alloc; i++)
				{
					printf("[gpu init ] ==%d<<%ld\n", i, _t->_mp_d[i]);
				}*/

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu decrypt ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu decrypt ] _t size%d\n", _t->_mp_size);
				gmp_printf("[gpu decrypt++] _t = %Zd\n", _t);

				///* load exp */
				printf("[gpu decrypt ] exp = %ld\n", _e[i]);

			}



		}

	}

	void gpu_decode(void** d_pub, void** d_plain, mpz_manager* h_mng)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int col, row, enc;

		/* load d_cipher */
		gpu_manager* h_plain;
		hipHostMalloc((void**)& h_plain, sizeof(gpu_manager));
		hipMemcpy(h_plain, *d_plain, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		col = h_plain->col;
		row = h_plain->row;
		enc = h_plain->ifenc;

		/* malloc h_mng */
		initManager_d(h_mng, col, row, BITS);
		h_mng->ifenc = 0;

		/* field dev2host */
		for (int i = 0; i < col * row; i++)
		{
			hipMemcpy(h_mng->field[i]->_mp_d, h_plain->d_field[i]._limbs, 
				nByte, hipMemcpyDeviceToHost);

			for (int j = h_mng->field[i]->_mp_alloc - 2; j >= 0; j--)
			{
				if (h_mng->field[i]->_mp_d[j] != (uint64_t)0)
				{
					h_mng->field[i]->_mp_size = j + 1;
					printf("++++++++++++-------%d\n", j);
					break;
				}
			}
		}

		/* exp dev2host */
		hipMemcpy(h_mng->exp, h_plain->d_exp, 
			col * row * sizeof(uint64_t), hipMemcpyDeviceToHost);
	}

	/* dump */
	int gpu_dump(void** d_mng, void** h_dumpMap)
	{
		int i, j;
		int nByte = BIT2BYTE(BITS);
		int limbs = nByte / sizeof(uint64_t);
		int col, row;
		int enc, mal;

		/* malloc h_contain */
		gpu_manager* h_mng;
		hipHostMalloc((void**)& h_mng, sizeof(gpu_manager));
		hipMemcpy(h_mng, *d_mng, sizeof(gpu_manager), hipMemcpyDeviceToHost);
		hipMemcpy(&col, &h_mng->col, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&row, &h_mng->row, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(&enc, &h_mng->ifenc, sizeof(int), hipMemcpyDeviceToHost);

		//printf("[gpu dump]:col = %d row = %d enc = %d\n", h_mng->col, h_mng->row, h_mng->ifenc);

		///* malloc byte */
		short headerSize = 4;
		long int dumpSize = (row * col * limbs + headerSize) + (row * col);
		*h_dumpMap = (uint64_t*)malloc(dumpSize * sizeof(uint64_t));

		/////* load header */
		uint64_t* iter = (uint64_t*)(*h_dumpMap);

		iter[0] = (uint64_t)row;
		iter[1] = (uint64_t)col;
		iter[2] = (uint64_t)BITS;
		iter[3] = (uint64_t)enc;

		///* load field */
		iter = ((uint64_t*)(*h_dumpMap)) + headerSize;
		hipMemcpy(iter, h_mng->d_field, nByte * col * row,
			hipMemcpyDeviceToHost);

		///* load exp */
		iter = ((uint64_t*)(*h_dumpMap)) + headerSize + row * col * limbs;
		hipMemcpy(iter, h_mng->d_exp, sizeof(uint64_t) * col * row,
			hipMemcpyDeviceToHost);

		if (0)
		{
			/* host memory */
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));

			/* fill value 2 host */
			hipMemcpy(_tmp, *d_mng, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t _e;

			for (i = 0; i < col * row; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}

				printf("[gpu dump ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu dump ] _t size%d\n", _t->_mp_size);
				gmp_printf("[...........gpu dump++]%Zd\n", _t);

				/* load exp */
				hipMemcpy(&_e, _tmp->d_exp, sizeof(uint64_t), hipMemcpyDeviceToHost);
				printf("[gpu dump ] exp = %ld\n", _e);
			}
		}


		return dumpSize;

	}

	int gpu_load(void** d_mng, uint64_t* h_data)
	{
		int i, j;
		int col, row, bit, enc;
		int nByte = BIT2BYTE(BITS);
		int limbs = nByte / sizeof(uint64_t);
		short headerSize = 4;

		/* load header */
		row = (int)h_data[0];
		col = (int)h_data[1];
		bit = (int)h_data[2];
		enc = (int)h_data[3];
		
		if (bit != BITS)
			return -1;

		/* device memcory */
		gpu_manager* _d_mng;
		hipMalloc((void**)& _d_mng, sizeof(gpu_manager));

		/* host memory */
		gpu_manager* h_contain;
		hipHostMalloc((void**)&h_contain, sizeof(gpu_manager));
		h_contain->col = col;
		h_contain->row = row;
		h_contain->ifenc = enc;
		h_contain->ifmal = 1;
		hipMalloc((void**)& h_contain->d_field, sizeof(gpu_mpz) * col * row);
		hipMalloc((void**)& h_contain->d_exp, sizeof(uint64_t) * col * row);
		

		printf("[gpu load] col = %d row = %d enc = %d bit = %d\n", 
			col, row, enc, bit);

		/* load field */
		uint64_t* iter ;
		for  (i = 0; i < col * row; i++)
		{
			iter = h_data + headerSize + i * limbs;
			hipMemcpy(h_contain->d_field[i]._limbs, iter, nByte,
				hipMemcpyHostToDevice);
		}

		/* load exp */
		iter = h_data + headerSize + row * col * limbs;
		hipMemcpy(h_contain->d_exp, iter, 
			sizeof(uint64_t) * col * row, hipMemcpyHostToDevice);	

		/* fill value 2 device */
		hipMemcpy(_d_mng, h_contain, sizeof(gpu_manager), hipMemcpyHostToDevice);

		/* fill addr 2 host */
		memcpy(d_mng, &_d_mng, sizeof(void*));

		hipHostFree(h_contain);
		
		if (0)
		{
			/* host memory */
			gpu_manager* _tmp;
			hipHostMalloc((void**)& _tmp, sizeof(gpu_manager));

			/* fill value 2 host */
			hipMemcpy(_tmp, *d_mng, sizeof(gpu_manager), hipMemcpyDeviceToHost);

			/* load base */
			mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, BITS);
			uint64_t _e;

			for (i = 0; i < col * row; i++)
			{
				hipMemcpy(_t->_mp_d, _tmp->d_field[i]._limbs, nByte, hipMemcpyDeviceToHost);

				for (int j = _t->_mp_alloc - 2; j >= 0; j--)
				{
					if (_t->_mp_d[j] != (uint64_t)0)
					{
						_t->_mp_size = j + 1;
						printf("++++++++++++-------%d\n", j);
						break;
					}
				}
				printf("[gpu load ]_t alloc %d\n", _t->_mp_alloc);
				printf("[gpu load ] _t size%d\n", _t->_mp_size);
				gmp_printf("[...........gpu load++]%Zd\n", _t);

				/* load exp */
				hipMemcpy(&_e, &(_tmp->d_exp[i]), sizeof(uint64_t), hipMemcpyDeviceToHost);
				printf("[gpu load ] exp = %ld\n", _e);
			}
		}

		return 1;
	}

	/* tool test*/
	__global__ void dev_pub_kernel(cgbn_public_key* pub)
	{
		printf("kernel here len++++++++++++++ \n");
		printf("pub.g = %d\n", pub->g._limbs[0]);
		printf("pub.n = %d\n", pub->n._limbs[0]);
		printf("pub.n2 = %d\n", pub->n2._limbs[0]);
		printf("pub.max = %d\n", pub->max_int._limbs[0]);


		int exp;
		double base = frexp(3.121441f, &exp);
		printf("3.121441 :exp = %d base = %lf\n", exp, base);

	}
	__global__ void dev_priv_kernel(cgbn_private_key* priv)
	{
		printf("kernel here len++++++++++++++ \n");
		printf("priv.p =  %d\n", priv->p._limbs[0]);
	}
	__global__ void dev_mpz_kernel(gpu_mpz* d_data)
	{
		
	}

	void gpu_show(void** dev_ptr, int n)
	{
		if (n == 0)
		{
			dev_pub_kernel << <1, 1 >> > ((cgbn_public_key*)(*dev_ptr));
			hipDeviceSynchronize();
		}
		
		if (n == 1)
		{
		/*	mpz_t _t;
			mpz_init(_t);
			mpz_setbit(_t, 2048);

			cgbn_public_key _p;
			hipMemcpy(&_p, *dev_ptr, sizeof(cgbn_public_key), hipMemcpyDeviceToHost);
			
			store2gmp(_t, &_p.g);*/
			dev_priv_kernel << <1, 1 >> > ((cgbn_private_key*)(*dev_ptr));
			hipDeviceSynchronize();
		}
		
		if (n == 2)
		{
			dev_mpz_kernel << <1, 1 >> > ((gpu_mpz*)(*dev_ptr));
			hipDeviceSynchronize();

		}

	}

}